#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>


__global__ void MulMat(float* mat1 , int alto1, int ancho1 , float* mat2 , int alto2 , int ancho2 , float* matriz){
	
	int i,j,n,pos;
  float valor;

  //Calculo de la posicion en X y Y  

  int o = blockIdx.y*blockDim.y+threadIdx.y;
  int m = blockIdx.x*blockDim.x+threadIdx.x;

  //Condicion que me dice cuando operar

  if ( o < alto1 && m < ancho2 ){
      //posicion de escritura
      
      pos = o*ancho1+m;
      
      //posicion en X para la lectura de la matriz1 
      
      i = o*ancho1;

      //posicion en Y para la lectura de la matriz2 
      
      j=m;

      //valor a escribir y contador del ciclo 
      
      valor=0;
      n=0;
      //ciclo que hace el producto punto 
          while(n<alto2){
              valor=valor + mat1[i]*mat2[j];
              i= i+1;
              j=j+ancho2;
              n=n+1;
              }

      //escritura del valor en la matriz
      matriz[pos]=valor;
  }
}

int main(int argc , char *argv[] ){

  if (argc != 3){
    printf("Uso: MultMat FileName1 FileName2\n" );
  }else{
    FILE *archivo1 , *archivo2 , *archivof;

    int a1valor1, a1valor2 , a2valor1 , a2valor2 , tamanom1 , tamanom2 ,tamanom3;

    float *h_matriz1 , *h_matriz2, *h_matrizf,*d_matriz1 , *d_matriz2, *d_matrizf;

    //se abre el file :v
    archivo1 = fopen ( argv[1], "r" );

    // se lee el file
    fscanf (archivo1, "%d", &a1valor1);
    fscanf (archivo1, "%d", &a1valor2);

    // se calcula el tamaño

    tamanom1 = a1valor1*a1valor2;

    //se crea la matriz plana!! (chan chan chaaaaan)

    h_matriz1 = (float*)malloc(tamanom1*sizeof(float));

    // se llena la matriz plana

    int n;
    n=0;

    while(n<tamanom1){
      float numero;
      fscanf (archivo1, "%f,", &numero);
      h_matriz1[n] = numero;
      n= n+1;
    }

    //se abre el file :v
    archivo2 = fopen ( argv[2], "r" );

    // se lee el file
    fscanf (archivo2, "%d", &a2valor1);
    fscanf (archivo2, "%d", &a2valor2);

    // se calcula el tamaño

    tamanom2 = a2valor1*a2valor2;

    //se crea la matriz plana!! (chan chan chaaaaan)


    h_matriz2 = (float*)malloc(tamanom2*sizeof(float));

    // se llena la matriz plana

    n=0;

    while(n<tamanom2){

      float numero;
      fscanf (archivo2, "%f,", &numero);
      h_matriz2[n] = numero;
      n= n+1;
    }

    tamanom3 = a1valor1 * a2valor2;

  //espacio necesario para las matrices en el device     
    size_t SizeMat1 = tamanom1*sizeof(float);
    size_t SizeMat2 = tamanom2*sizeof(float);
    size_t SizeMat3 = tamanom3*sizeof(float);
 
  //reserva de memoria en el device 

    hipMalloc(&d_matriz1, SizeMat1);
    hipMalloc(&d_matriz2, SizeMat2);
    hipMalloc(&d_matrizf, SizeMat3);
    h_matrizf = (float*)malloc(tamanom3*sizeof(float));

  //copia de vectores al device 

    hipMemcpy( d_matriz1, h_matriz1, SizeMat1, hipMemcpyHostToDevice);  
    hipMemcpy( d_matriz2, h_matriz2, SizeMat2, hipMemcpyHostToDevice);

  //calculo de la cantidad del tamaño del grid si cada bloque tiene 1024 hilos en X y Y 

    int blockSize , gridSizeX , gridSizeY ;

    //Numero de hilos por bloque 
    
    blocksize = 32; // 32*32 = 1024 donde  1024 es la cantidad de hilos por bloque! 

    //Numero de bloques en X y Y 

    gridSizeX = (int)ceil((float)a2valor2/blockSize);
    gridSizeY = (int)ceil((float)a1valor1/blockSize); 

    //creacion de las dimenciones para el kernel 

    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimGrid(gridSizeX,gridSizeY,1);

    //Cambiar por copia de matris final en dispositivo   
    MulMat<<<dimGrid,dimBlock>>>(d_matriz1,a1valor1,a1valor2,d_matriz2,a2valor1,a2valor2,d_matrizf);

    //Copiar resultado al host 
    hipMemcpy( h_matrizf, d_matrizf,SizeMat3, hipMemcpyDeviceToHost );


  //se cierran los archivoss de las matrices

	fclose(archivo1);
	fclose(archivo2);
 
  //se libera memoria en el host 

	free(h_matriz1);
  free(h_matriz2);

  //se libera memoria en el device 

  hipFree(d_matriz1);
  hipFree(d_matriz2);
  hipFree(d_matrizf);

  //escritura en archivo de salida 

	archivof = fopen ( "out.txt", "w" );

	int tm;
	tm = a1valor1*a2valor2;

	fprintf(archivof , "%d\n%d\n" , a1valor1 , a2valor2 );

	int i;

	i=0;

	while(i<tm){

		if(i%a2valor2 == 0){
			fprintf(archivof , "\n");
		}
		
    fprintf(archivof , "%.0f," , h_matrizf[i]);
		i=i+1;
	}

  //se termina de liberar memoria

  free(h_matrizf);

  }

  return 0;
}
